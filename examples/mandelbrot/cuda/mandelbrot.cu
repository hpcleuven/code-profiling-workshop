#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define MAXITER 255

int get_N(int argc, char *argv[], int* blocksize) {
    /* Read an integer as the (only) command-line argument */
    if (argc != 3) {
        printf("Expected exactly 2 arguments: N, blocksize\n");
        exit(EXIT_FAILURE);
    }

    int N = atoi(argv[1]);

    *blocksize = atoi(argv[2]);

    if (N % *blocksize != 0) {
        printf("blocksize should evenly divide N\n");
        exit(EXIT_FAILURE);
    }

    return N;
}

__global__ void mandelbrot_kernel(int N, int Nx, int Ny, int *arr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    double x = (1.0 * i - 2.0 * N) / N;
    double y = (1.0 * j - 1.0 * N) / N;

    double wx = 0;
    double wy = 0;
    double v = 0;
    double xx = 0;
    int k;

    for (k = 0; k < MAXITER; k++){
        xx = wx*wx - wy*wy;
        wy = 2.0*wx*wy;
        wx = xx + x;
        wy = wy + y;
        v = wx*wx + wy*wy;
        if (v >= 4.0) break;
    }
    arr[i*Ny + j] = k;
}

int main(int argc, char *argv[]) {
    // Start a timer
    struct timeval begin, end;
    gettimeofday(&begin, 0);

    // Check input parameters
    int blocksize;
    int N = get_N(argc, argv, &blocksize);

    // Compute local size and offset
    int myNx = 3 * N;
    int myNy = 2 * N;

    // Initialize result array
    int *arr_dev = NULL;
    int err = hipMalloc(&arr_dev, sizeof(int) * myNx * myNy);
    if (err != 0) {
        printf("hipMalloc failed!\n");
        return err;
    }

    // Do the actual work
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(myNx/blocksize, myNy/blocksize);
    mandelbrot_kernel<<<dimGrid, dimBlock>>>( N, myNx, myNy, arr_dev);

    // Copy data from device to host for postprocessing
    int *arr = (int *)malloc(sizeof(int) * myNx * myNy);
    hipMemcpy(arr, arr_dev, sizeof(int) * myNx * myNy, hipMemcpyDeviceToHost);

    long long int niter = 0;
    for (int k=0; k < myNx * myNy; k++) niter += arr[k];

    // Write output to file
    FILE *fh = fopen("output.bin", "wb");
    fwrite(&arr[0], sizeof(int), myNx * myNy, fh);
    fclose(fh);

    // Stop measuring time and calculate the elapsed time
    gettimeofday(&end, 0);
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
    double elapsed = seconds + microseconds*1e-6;    
    printf("Performed %16lld iterations in %8.3fs\n", niter, elapsed);

    // Finalize
    free(arr);
    return EXIT_SUCCESS;
}
